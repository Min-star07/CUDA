#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
// Initilization data on host
void InitiData(float *vec, int size){
    for(int i =0; i < size; i++){
        vec[i] = rand()/(float)RAND_MAX;
        // printf("%d, %.1f\n", i, vec[i]);
    }
}

// sum on host
void Sum_vec_on_CPU(float * vec1, float *vec2, float *vec3, int size){
    for(int i = 0; i < size; i ++){
        vec3[i] = vec1[i] + vec2[i];
    }
}

// compare the difference between CPU and GPU
void CheckResult(float *CPU_res, float *GPU_res, int size){
    float error_bar = 1e-6;
    for(int i =0; i < size; i++){
        if((CPU_res[i] - GPU_res[i] ) > error_bar){
            printf("ThE result is diffeence between GPU (%d, %.1f) and CPU (%d, %.1f)\n", i, CPU_res[i], i , GPU_res[i]);
            exit(1);
        }
    }
}

// CUDA ERROR CHECK
#define cudaCheckErrors(msg){\
    hipError_t __err = hipGetLastError();\
    if(__err != hipSuccess){\
    fprintf(stderr, "Fatal error: %s ===> %s ===>  %s : %d\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);\
    fprintf(stderr, "FURTHER CHECK *****  ABORTION\n");\
    exit(1);\
};\
}

// kernal
__global__ void Sum_vec_on_GPU(float *vec1, float *vec2, float *vec3, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size){
        vec3[idx] = vec1[idx] + vec2[idx];
        // printf("%d, %d, %d, %d, %d, %.1f\n",  idx, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x, vec3[idx]);
    }
    __syncthreads();
}

void setGPU(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount  < 1){
        printf("No CUDA device found, exiting...\n");
        exit(1);
    }
    else{
        for(int i = 0; i < deviceCount; i++ ){
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
             printf("Device %d: %s has compute capability : Major: %d Minor: %d \n", i, deviceProp.name, deviceProp.major, deviceProp.minor);

            // set GPU
            hipSetDevice(i);
            printf("Set GPU %d.\n", i);
        }

    }
}

double timeCount(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec *1.e-6);
}

int main(int argc, char** argv){
    // set GPU
    setGPU();
    cudaCheckErrors("set GPU failed");
    // define varilable on host and device
    float *h_a, *h_b, *GPU_res,*CPU_res;
    float *d_a, *d_b, *d_c;
    const int DSIZE = 1 << 21;
    size_t nBytes = DSIZE * sizeof(float);

    // allocate zerocopy memory
    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc((void**)&h_a, nBytes, flags);
    hipHostAlloc((void**)&h_b, nBytes, flags);

    // initi data on host
    InitiData(h_a, DSIZE);
    InitiData(h_b, DSIZE);

    GPU_res = (float*)malloc(nBytes);
    CPU_res = (float*)malloc(nBytes);
    memset(CPU_res, 0, nBytes);
    memset(GPU_res, 0, nBytes);

    //pass the pointer to device
    hipHostGetDevicePointer((void**)&d_a, (void*)h_a, 0);
    hipHostGetDevicePointer((void**)&d_b, (void*)h_b, 0);
    hipMalloc((void**)&d_c, nBytes);
    cudaCheckErrors("Alloc zero memory failed");

    // Get the result oon the CPU;
    double iStart, iElaps;
    iStart = timeCount();
    Sum_vec_on_CPU(h_a, h_b, CPU_res, DSIZE);
    iElaps = timeCount() - iStart;
    printf("Time taken on CPU : %f seconds.\n", iElaps);

    // warming up
    dim3 block(512);
    dim3 grid((DSIZE + block.x -1)/ block.x); 
    iStart = timeCount();
    Sum_vec_on_GPU<<<grid, block>>>(d_a, d_b, d_c, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel during warming up : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch warming up failure");

    // launch kernel
    iStart = timeCount();
    Sum_vec_on_GPU<<<grid, block>>>(d_a, d_b, d_c, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch failure");
    // copy result from device to host
    hipMemcpy(GPU_res, d_c, nBytes, hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    // check result
    CheckResult(CPU_res, GPU_res, DSIZE);
    printf("Success!\n"); 
    hipHostFree(h_a);
    hipHostFree(h_b);
    free(CPU_res);
    free(GPU_res);
    // cudaFree(d_a);  device fopy memory in the host, not alloc so do not cudafree them
    // cudaFree(d_b);
    hipFree(d_c);

    return 0;
}


