#include <hip/hip_runtime.h>
#include <stdio.h>
void checkresult(float *CPU_RESULT, float *GPU_RESULT, int size){
    float error = 1e-6;
    printf("%d" , size);
    for (int i = 0; i < size; i++)
    {
        float diff = CPU_RESULT[i] - GPU_RESULT[i];
        if(diff < -error || diff > error){
            printf("mismatch %d CPU_result : %.1f, GPU_result %.1f\n ", i, CPU_RESULT[i], GPU_RESULT[i]);
            exit(-1);
        }
        // else{
        //     printf("successful reulst \n");
        // }
    }
}
void addarrayonCPU(float *A, float *B, float *cpu_result, int size){
    for(int i = 0; i < size; i++){
        cpu_result[i] = A[i] + B[i];
    }
}

void initilization(float *before, float*after, int size){
    for(int i = 0; i < size; i++){
        before[i] = 1.0f;
        after[i] = 2.0f;
    }
}

__global__ void addarrayonGPU(float *a, float *b, float *c, int size){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    ;
    if(idx < size){
        c[idx] = a[idx] + b[idx];
        // printf("idx = %d, Result = %.1f\n", idx, c[idx]);
    }
}
int main(int argc, char **argv){
    // Set up device
    int dev = 0;
    hipSetDevice(dev);

    // Getdeviceproperties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    // check if support mapped memory
    if(!deviceProp.canMapHostMemory){
        printf("Device does not support mapped memory\n");
        return 1;
    }
    printf("Using Device %d : %s\n", dev, deviceProp.name);

    // set up vector size
    int ipower = 5;
    if(argc >1) ipower =atoi(argv[1]);
    int size = 1 << ipower;
    if(ipower <18)
        printf("memory size : %1.f kb\n", size * sizeof(float)/ 1024.0);
    else{
        printf("memory size : %1.f mb\n", size * sizeof(float) / (1024.0 * 1024.0));
    }

    // copy from host memory
    float *h_a, *h_b, *h_c, *CPURes;
    float *d_a, *d_b, *d_c;
    // int size = 1 << 24;
    size_t nbytes = size * sizeof(float);


    dim3 block(32);
    dim3 grid((size + block.x - 1) / block.x);

    // allocate the host memory void* malloc(size_t size);

    h_a = (float*) malloc(nbytes);
    h_b = (float*) malloc(nbytes);
    h_c = (float*) malloc(nbytes);
    CPURes = (float*) malloc(nbytes);

    // // initilizartion data
    // initilization(h_a, h_b, size);

    // allocate the device memory cudaError_t cudaMalloc(void** devPtr, size_t size);
    hipMalloc((float **)&d_a, nbytes);
    hipMalloc((float **)&d_b, nbytes);
    hipMalloc((float **)&d_c, nbytes);

    // copy memory form host to device
    hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, nbytes, hipMemcpyHostToDevice);

     // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);
    addarrayonGPU<<<grid, block>>>(d_a, d_b, d_c, size);
    // cudaDeviceSynchronize();
     // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the time taken by the kernel execution
    printf("Time taken by kernel: %f ms\n", milliseconds);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_c, d_c, nbytes, hipMemcpyDeviceToHost);

    // check the result
    // checkresult(h_a, h_c, size)
    addarrayonCPU(h_a, h_b, CPURes, size);
    checkresult(CPURes, h_c, size);

    // free the memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // usinf zerocopy memory fro arrayA and B;
    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc((float **)&h_a, nbytes, flags);
    hipHostAlloc((float **)&h_b, nbytes, flags); //
    hipHostAlloc((float **)&h_c, nbytes, flags); //

    // initialize data at host side
    initilization(h_a, h_b, size);

    // copy the data from device to host
    // cudaMemcpy(h_a, d_a, nbytes, cudaMemcpyDeviceToHost);
    memset(CPURes, 0, nbytes);
    memset(h_c, 0, nbytes);

    // pass the pointer to device
    hipHostGetDevicePointer((float **)&d_a, (void *)h_a, 0);
    hipHostGetDevicePointer((float **)&d_b, (void *)h_b, 0);
    hipHostGetDevicePointer((float **)&d_c, (void *)h_c, 0);

    // add on cpu
    addarrayonCPU(h_a, h_b, CPURes, size);
    
    // launch kernel
    // Create CUDA events
    // cudaEvent_t start_zerocopy, stop_zerocopy;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);
    addarrayonGPU<<<grid, block>>>(d_a, d_b, d_c, size);
    // cudaDeviceSynchronize();
     // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    // float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the time taken by the kernel execution
    printf("Time taken by kernel: %f ms\n", milliseconds);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);


    hipMemcpy(h_c, d_c, nbytes, hipMemcpyDeviceToHost);

    // check the result
    checkresult(CPURes, h_c, size);

    // free the memory
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    free(CPURes);

    return 0;
}