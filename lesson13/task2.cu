#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

using namespace cooperative_groups;

const int nTPB = 256;
__device__ int reduce(thread_group g, int *x, int val){
    int lane = g.thread_rank();
    
       
    for(int i = g.size()/2; i>0;i /=2){
        x[lane] = val;
        printf("Hello from g3_rank 0: %lu %d, %d\n",g.thread_rank(), lane, val);
        g.sync();
        if(lane<i)
            val += x[lane+i];
            g.sync();
    }
      if (g.thread_rank() == 0) printf("group partial sum: %d\n", val);
  return val;
}

__global__ void my_reduce_kernel(int *data){
    __shared__ int sdata[nTPB];

    // task 1a: create a proper thread block group below
    auto g1 = this_thread_block();
    size_t gindex = g1.group_index().x *nTPB + g1.thread_index().x;

    // task 1b: uncomment and create a proper 32-thread tile below, using group g1 created above
    auto g2 = tiled_partition(g1, 32);
    
    // task 1c: uncomment and create a proper 16-thread tile below, using group g2 created above
    auto g3 = tiled_partition(g2, 16);
    // size_t g3index = g3.group_index().x *nTPB + g3.thread_index().x;
    // for each task, adjust the group to point to the last group created above
    auto g = g3;

    // Make sure we send in the appropriate patch of shared memory
    int sdata_offset = (g1.thread_index().x / g.size()) *g.size();
    // printf("index : %u, gindex : %lu, threads : %d\n", g1.group_index().x, gindex, g1.thread_index().x);
    reduce(g, sdata + sdata_offset, data[gindex]);
    // if (g3.thread_rank()==0)
    // printf("Hello from g3 rank 0: %d\n",
    //    g1.thread_rank());
    // printf("Block index: %d, Thread index: %d, Global index (gindex): %lu\n",
    //    g.group_index().x, g1.thread_index().x, gindex);
}

int main(){
    int *data;
    hipMallocManaged(&data, nTPB*sizeof(data[0]));
    for (int i = 0; i < nTPB; i++) data[i] = i;
    my_reduce_kernel<<<1,nTPB>>>(data);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("cuda error: %s\n", hipGetErrorString(err));
}