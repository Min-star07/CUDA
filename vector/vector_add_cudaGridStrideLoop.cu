#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
// Initilization data on host
void InitiData(float *vec, int size){
    for(int i =0; i < size; i++){
        vec[i] = rand()/(float)RAND_MAX;
        // printf("%d, %.1f\n", i, vec[i]);
    }
}

// sum on host
void Sum_vec_on_CPU(float * vec1, float *vec2, float *vec3, int size){
    for(int i = 0; i < size; i ++){
        vec3[i] = vec1[i] + vec2[i];
    }
}

// compare the difference between CPU and GPU
void CheckResult(float *CPU_res, float *GPU_res, int size){
    float error_bar = 1e-6;
    for(int i =0; i < size; i++){
        if((CPU_res[i] - GPU_res[i] ) > error_bar){
            printf("ThE result is diffeence between GPU (%d, %.1f) and CPU (%d, %.1f)\n", i, CPU_res[i], i , GPU_res[i]);
            exit(1);
        }
    }
}

// CUDA ERROR CHECK
#define cudaCheckErrors(msg){\
    hipError_t __err = hipGetLastError();\
    if(__err != hipSuccess){\
    fprintf(stderr, "Fatal error: %s ===> %s ===>  %s : %d\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);\
    fprintf(stderr, "FURTHER CHECK *****  ABORTION\n");\
    exit(1);\
};\
}

// kernal
__global__ void Sum_vec_on_GPU_normal(float *vec1, float *vec2, float *vec3, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size){
        vec3[idx] = vec1[idx] + vec2[idx];
        // printf("%d, %d, %d, %d, %d, %.1f\n",  idx, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x, vec3[idx]);
    }
    __syncthreads();
}

// kernal grid stride loop
__global__ void Sum_vec_on_GPU_loop(float *vec1, float *vec2, float *vec3, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
   for(int i = idx; i < size; i += stride){
        vec3[i] = vec1[i] + vec2[i];
        // printf("%d, %d, %d, %d, %d, %.1f\n",  idx, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x, vec3[idx]);
    }
    __syncthreads();
}

void setGPU(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount  < 1){
        printf("No CUDA device found, exiting...\n");
        exit(1);
    }
    else{
        for(int i = 0; i < deviceCount; i++ ){
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
             printf("Device %d: %s has compute capability : Major: %d Minor: %d \n", i, deviceProp.name, deviceProp.major, deviceProp.minor);
            // set GPU
            hipSetDevice(i);
            printf("Set GPU %d.\n", i);
        }
    }
}

double timeCount(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec *1.e-6);
}

int main(int argc, char** argv){
    // set GPU
    setGPU();
    cudaCheckErrors("set GPU failed");
    // define varilable on host and device
    float *A, *B, *GPU_res, *CPU_res;
    const int DSIZE = 1 << 21;
    size_t nBytes = DSIZE * sizeof(float);
    hipMallocManaged((void **)&A, nBytes);
    hipMallocManaged((void **)&B, nBytes);
    hipMallocManaged((void **)&GPU_res, nBytes);
    hipMallocManaged((void **)&CPU_res, nBytes);

    // initi data on host
    InitiData(A, DSIZE);
    InitiData(B, DSIZE);
    memset(CPU_res, 0, nBytes);
    memset(GPU_res, 0, nBytes);

    // Count time;
    // Get the result oon the CPU;
    double iStart, iElaps;
    iStart = timeCount();
    Sum_vec_on_CPU(A, B, CPU_res, DSIZE);
    iElaps = timeCount() - iStart;
    printf("Time taken on CPU : %f seconds.\n", iElaps);

    // warming up
    dim3 block(512);
    dim3 grid((DSIZE + block.x -1)/ block.x); 
    iStart = timeCount();
    Sum_vec_on_GPU_normal<<<grid, block>>>(A , B, GPU_res, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel during warming up : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch warming up failure");

    // launch kernel
    iStart = timeCount();
    Sum_vec_on_GPU_normal<<<grid, block>>>(A , B, GPU_res, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel by normal mode : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch failure");

     // launch kernel
    iStart = timeCount();
    Sum_vec_on_GPU_loop<<<grid, block>>>(A , B, GPU_res, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel by srid stride loop mode : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch failure");

    // check result
    CheckResult(CPU_res, GPU_res, DSIZE);
    printf("Success!\n"); 
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(GPU_res);
    hipHostFree(CPU_res);

    return 0;
}


