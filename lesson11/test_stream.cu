
#include <hip/hip_runtime.h>
#include<math.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>

typedef float ft;
const int chunks = 64;
const size_t ds = 1024*1204*chunks;
const int count = 22;
const int num_streams =8;


// not modifiable
const float sqrt_2PIf = 2.5066282747946493232942230134974f;
const double sqrt_2PI = 2.5066282747946493232942230134974;

__device__ float gpdf(float val, float sigma){
    return expf(-0.5f*val*val) /(sigma *sqrt_2PIf);
}

__device__ double gpdf(double val, double sigma){
    return expf(-0.5f*val*val) /(sigma *sqrt_2PIf);
}

__global__ void gaussian_pdf(const ft * __restrict__ x, ft * __restrict__ y, const ft mean, const ft sigma, const int n){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < n){
        ft in = x[idx] - (count/2)*0.01f;
        ft out = 0;
        for(int i =0; i< count;i++){
            ft temp = (in - mean)/sigma;
            out += gpdf(temp, sigma);
            in += 0.01f;
        }
        y[idx] = out / count;
    }
}

#define cudaCheckErrors(msg)\
do{\
    hipError_t __err = hipGetLastError();\
    if(__err != hipSuccess){\
        fprintf(stderr, "Fatal error :  %s : (%s at %s : %d)\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);\
        fprintf(stderr, "***FAILD - ABORTING \n");\
        exit(1);\
    }\
}while(0)

// host-based timing
#define USECPSEC 1000000ULL
unsigned long long dtime_usec(unsigned long long start){
    timeval tv;
    gettimeofday(&tv, 0);
    return ((tv.tv_sec*USECPSEC) +tv.tv_usec) - start;
} 

int main(){
    ft *h_x, *d_x, *h_y, *h_y1, *d_y;
    hipHostAlloc(&h_x, ds *sizeof(ft), hipHostMallocDefault);
    hipHostAlloc(&h_y, ds *sizeof(ft), hipHostMallocDefault);
    hipHostAlloc(&h_y1, ds *sizeof(ft), hipHostMallocDefault);
    hipMalloc(&d_x, ds *sizeof(ft));
    hipMalloc(&d_y, ds*sizeof(ft));
    cudaCheckErrors("allocation error");

    hipStream_t streams[num_streams];
    for(int i =0; i < num_streams; i++){
        hipStreamCreate(&streams[i]);
    }
    cudaCheckErrors("stream creation error");

    gaussian_pdf<<<(ds+255)/256, 256>>>(d_x, d_y, 0.0, 1.0, ds);
    for(size_t i = 0; i < ds; i++ ){
        h_x[i] = rand()/(ft) RAND_MAX;
    }

    hipDeviceSynchronize();
    unsigned long long et1 = dtime_usec(0);
    hipMemcpy(d_x, h_x, ds*sizeof(ft), hipMemcpyHostToDevice);
    gaussian_pdf<<<(ds+255)/256, 256>>>(d_x, d_y, 0.0, 1.0, ds);
    hipMemcpy(h_y1, d_y, ds*sizeof(ft), hipMemcpyDeviceToHost);
    cudaCheckErrors("non-streams execution error");

    et1 = dtime_usec(et1);
    std::cout << "non-stream elapsed time : " << et1/(float)USECPSEC << std::endl;
#ifdef USE_STREAMS
    cudaMemset(d_y, 0, ds * sizeof(ft));

    unsigned long long et = dtime_usec(0);

    for(int i =0; i < chunks; i++){
        cudaMemcpyAsync(d_x + i * (ds/chunks) , h_x + i *(ds/chunks), (ds/chunks) * sizeof(ft), cudaMemcpyHostToDevice, streams[i %num_streams]);
        gaussian_pdf<<<(ds/chunks) +255/ 256, 256, 0, streams[i%num_streams]>>>(d_x + i*(ds/chunks), d_y +i *(ds/chunks), 0.0, 1.0, ds/chunks);
        cudaMemcpyAsync(h_y + i *(ds/chunks), d_y + i * (ds/chunks), (ds/chunks) * sizeof(ft), cudaMemcpyDeviceToHost, streams[i%num_streams]);
    }
    cudaDeviceSynchronize();
    cudaCheckErrors("streams execution error");
    et = dtime_usec(et);
    for (int i = 0; i < ds; i++) {
    if (h_y[i] != h_y1[i]) {
      std::cout << "mismatch at " << i << " was: " << h_y[i] << " should be: " << h_y1[i] << std::endl;
      return -1;
    }
  }

  std::cout << "streams elapsed time: " << et/(float)USECPSEC << std::endl;
#endif

    return 0;
}