#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const int DSIZE = 4;
const int block_size = 2;  // CUDA maximum is 1024 *total* threads in block
const float A_val = 3.0f;
const float B_val = 2.0f;

__global__ void mmul(const float *A, const float *B, float *C, int ds){
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // column index
    int idy = threadIdx.y + blockDim.y * blockIdx.y; // row index
    
    // Check bounds
    if((idx < ds) && (idy < ds)){
        float temp = 0;
        for(int i = 0; i < ds; i++){
            temp += A[idy * ds + i] * B[i * ds + idx];
        }
        C[idy * ds + idx] = temp;
    }
}

int main(int argc, char **argv){
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // Timing variables
    clock_t t0, t1, t2;
    double t1sum = 0;
    double t2sum = 0;

    // Start timing
    t0 = clock();
    h_A = new float[DSIZE * DSIZE];
    h_B = new float[DSIZE * DSIZE];
    h_C = new float[DSIZE * DSIZE];

    for(int i = 0; i < DSIZE * DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
    }

    // Initialization timing
    t1 = clock();
    t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("Init took %f seconds, Begin compute \n", t1sum);

    // Allocate device memory and copy input data to GPU
    hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * DSIZE * sizeof(float));

    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
    
    cudaCheckErrors("hipMemcpy H2D failure");

    dim3 block(block_size, block_size);
    dim3 grid((DSIZE + block.x - 1) / block.x, (DSIZE + block.y - 1) / block.y);

    mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);

    cudaCheckErrors("kernel launch failure");

    hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    
    // GPU timing
    t2 = clock();
    t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
    printf("Done. Compute took %f seconds\n", t2sum);

    // Verify result
    cudaCheckErrors("hipMemcpy D2H failure");
    for(int i = 0; i < DSIZE * DSIZE; i++){
        if(h_C[i] != A_val * B_val * DSIZE) {
            printf("Mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val * B_val * DSIZE);
            return -1;
        }
    }

    printf("Success!\n");

    // Free allocated memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
