#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#define CHECK(call)                                                                \
{                                                                                  \
        const hipError_t error = call;                                            \
        if (error != hipSuccess)                                                  \
        {                                                                          \
            printf("Error : %s : %d", __FILE__, __LINE__);                         \
            printf("code : %d,  reason : %s\n", error, hipGetErrorString(error)); \
            exit(1);                                                               \
        }                                                                          \
}

void checResult(float *cpuRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = 1;
    for(int i = 0; i < N; i++){
        if(fabs(cpuRef[i] - gpuRef[i]) > epsilon){
            printf("Mismatch at index %d, CPU: %5.2f, GPU: %5.2f\n", i, cpuRef[i], gpuRef[i]);
            match = 0;
            break;
        }
    }
    if(match )
    printf("Results match!\n");
}

// void initializeData(float * ip, int size){
//     // generate different seed for random number
//     unsigned int seed = time(NULL);
//     for(int i = 0; i < size; i++){
//         ip[i] = rand_r(&seed) / ((float)RAND_MAX);
//     }
// }
void initializeData(float * ip, int size){
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    for(int i = 0; i < size; i++){
        ip[i] = (float) (rand() &0xFF) / 10.0f;
    }
}

void sumArraysOnCPU(float * A, float *B, float *C, const int N){
    for(int i = 0; i < N; i++){
        C[i] = A[i] + B[i];
    }
}

__global__ void sumArrayOnGPU(float * A, float * B, float *C, const int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec *1.e-6);
}
int main(int argc, char ** argv){
    printf("%s Starting ....\n", argv[0]);
    //1. set up device
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
    CHECK(hipGetDeviceCount(&deviceCount));
    printf("Found %d CUDA device(s)\n", deviceCount);
    if(deviceCount <1){
        printf("No CUDA device found, exiting...\n");
        return 1;
    }
    else{
        printf("Found %d CUDA campatable GPU(s) in your computer.\n", deviceCount);
        for(int device = 0; device < deviceCount; device++){
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, device);
            printf("Device %d: %s has compute capability : Major: %d Minor: %d \n", device, deviceProp.name, deviceProp.major, deviceProp.minor);

            // set GPU
            CHECK(hipSetDevice(device));
            printf("Set GPU %d.\n", device);
    }
    }
    //2. set up data size of vectors
    const int size = 1 << 24;
    printf("Vectors of size %d.\n", size);
    //3. allocate memory on host and device
    float *h_A, *h_B;
    float *d_A, *d_B, *d_C;
    float *CPURef, *GPURef;
    // 3.1 allocate memory on host
    size_t nBytes = size * sizeof(float);
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    CPURef = (float *)malloc(nBytes);
    GPURef = (float *)malloc(nBytes);
    double iStart, iElaps;
    // 3.2 allocate memory on device
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));
    // 3.3 initialize data
    iStart = cpuSecond();
    initializeData(h_A, size);
    initializeData(h_B, size);
    iElaps = cpuSecond() - iStart;
    printf("Initializing data on host took %f seconds.\n", iElaps);
    memset(CPURef, 0, nBytes);
    memset(GPURef, 0, nBytes);
    // 4. sum arrays on CPU
    iStart = cpuSecond();
    sumArraysOnCPU(h_A, h_B, CPURef, size);
     iElaps = cpuSecond() - iStart;
    printf("Time elapsed on CPU took %f seconds.\n", iElaps);
    // 4. copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    // 5. launch kernal
    // dim3 block (size);
    // dim3 grid((size + block.x - 1) / block.x);
    dim3 block (256);
    dim3 grid((size + block.x - 1) / block.x);
    // 5.1 time to launch kernal
    iStart = cpuSecond();
    sumArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("Execution configration : <<<%d, %d>>> Time elapsed on GPU  %f seconds\n", grid.x, block.x, iElaps);
    // 6. copy data from device to host
    CHECK(hipMemcpy(GPURef, d_C, nBytes, hipMemcpyDeviceToHost));
    // 7. check results
    checResult(CPURef, GPURef, size);
    // 8. free memory on host and device
    free(h_A);
    free(h_B);
    free(CPURef);
    free(GPURef);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    printf("%s Done.\n", argv[0]);
    return 0;
}