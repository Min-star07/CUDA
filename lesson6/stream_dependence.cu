#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include <omp.h>
#include "../freshman.h"
#define N 300000
__global__ void kernel_1()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_2()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_3()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_4()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}

int main(){
    // setenv("CUDA_DEVICE_MAX_CONNECTIONS","4",1);
    setGPU();
    int n_stream = 5;
     hipStream_t *stream=(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
    for(int i=0;i<n_stream;i++)
    {
        hipStreamCreate(&stream[i]);
    }
    
    dim3 block(1);
    dim3 grid(1);

    hipEvent_t *event = (hipEvent_t*)malloc(n_stream*sizeof(hipEvent_t));
    for(int i =0; i < n_stream; i++){
        hipEventCreateWithFlags(&event[i], hipEventDisableTiming);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i = 0; i < n_stream; i++){
        kernel_1<<<grid, block, 0, stream[i]>>>();
        kernel_2<<<grid, block, 0, stream[i]>>>();
        kernel_3<<<grid, block, 0, stream[i]>>>();
        kernel_4<<<grid, block, 0, stream[i]>>>();
        hipEventRecord(event[i], stream[i]);
        hipStreamWaitEvent(stream[n_stream-1], event[i], 0);
    }
    hipEventRecord(stop, 0);

    // hipDeviceSynchronize(stop);
    CHECK(hipEventSynchronize(stop));

    float elapsed_time;

    // cudaEventElapsed(&elapsed_time, stop);
    hipEventElapsedTime(&elapsed_time,start,stop);

    printf("elapsed time:%f ms\n",elapsed_time);

    for(int i =0; i < n_stream; i++){
        // cudaStreamDestory(stream[i]);
        hipStreamDestroy(stream[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(stream);

    CHECK(hipDeviceReset());

    return 0;
}