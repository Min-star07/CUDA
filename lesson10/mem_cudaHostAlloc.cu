#include <cstdio>
#include <hip/hip_runtime.h>

// CUDA error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 100;
    size_t size = N * sizeof(int);

    // Pinned host memory allocation with mapping
    int *h_a, *h_b, *h_c;
    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc(&h_a, size, flags);  // Pinned memory on the host
    hipHostAlloc(&h_b, size, flags);
    hipHostAlloc(&h_c, size, flags);

    // Initialize host arrays with values
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Get device pointers to mapped host memory
    int *d_a, *d_b, *d_c;
    hipHostGetDevicePointer(&d_a, h_a, 0);  // d_a points to h_a
    hipHostGetDevicePointer(&d_b, h_b, 0);  // d_b points to h_b
    hipHostGetDevicePointer(&d_c, h_c, 0);  // d_c points to h_c

    // Launch kernel for vector addition
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);
    cudaCheckErrors("Kernel launch failed");

    // Wait for kernel to complete before accessing results on the host
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel execution failed");

    // Since the host memory is mapped to the device, no need for cudaMemcpy

    // Print results
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free pinned host memory (automatically mapped to the device)
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}
