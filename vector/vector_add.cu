#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
// Initilization data on host
void InitiData(float *vec, int size){
    for(int i =0; i < size; i++){
        vec[i] = rand()/(float)RAND_MAX;
        // printf("%d, %.1f\n", i, vec[i]);
    }
}

// sum on host
void Sum_vec_on_CPU(float * vec1, float *vec2, float *vec3, int size){
    for(int i = 0; i < size; i ++){
        vec3[i] = vec1[i] + vec2[i];
    }
}

// compare the difference between CPU and GPU
void CheckResult(float *CPU_res, float *GPU_res, int size){
    float error_bar = 1e-6;
    for(int i =0; i < size; i++){
        if((CPU_res[i] - GPU_res[i] ) > error_bar){
            printf("ThE result is diffeence between GPU (%d, %.1f) and CPU (%d, %.1f)\n", i, CPU_res[i], i , GPU_res[i]);
            exit(1);
        }
    }
}

// CUDA ERROR CHECK
#define cudaCheckErrors(msg){\
    hipError_t __err = hipGetLastError();\
    if(__err != hipSuccess){\
    fprintf(stderr, "Fatal error: %s ===> %s ===>  %s : %d\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);\
    fprintf(stderr, "FURTHER CHECK *****  ABORTION\n");\
    exit(1);\
};\
}

// kernal
__global__ void Sum_vec_on_GPU(float *vec1, float *vec2, float *vec3, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < size){
        vec3[idx] = vec1[idx] + vec2[idx];
        // printf("%d, %d, %d, %d, %d, %.1f\n",  idx, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x, vec3[idx]);
    }
    __syncthreads();
}

void setGPU(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount  < 1){
        printf("No CUDA device found, exiting...\n");
        exit(1);
    }
    else{
        for(int i = 0; i < deviceCount; i++ ){
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
             printf("Device %d: %s has compute capability : Major: %d Minor: %d \n", i, deviceProp.name, deviceProp.major, deviceProp.minor);

            // set GPU
            hipSetDevice(i);
            printf("Set GPU %d.\n", i);
        }

    }


}


double timeCount(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec *1.e-6);
}


int main(int argc, char** argv){
    // set GPU
    setGPU();
    cudaCheckErrors("set GPU failed");
    // define varilable on host and device
    float *h_a, *h_b, *GPU_res,*CPU_res;
    float *d_a, *d_b, *d_c;
    const int DSIZE = 1 << 21;
    size_t nBytes = DSIZE * sizeof(float);

    // allocate memory in the host
    h_a = (float*)malloc(nBytes);
    h_b = (float*)malloc(nBytes);
    GPU_res = (float*)malloc(nBytes);
    CPU_res = (float*)malloc(nBytes);

    // initi data on host
    InitiData(h_a, DSIZE);
    InitiData(h_b, DSIZE);
    memset(CPU_res, 0, nBytes);
    memset(GPU_res, 0, nBytes);
    // Get the result oon the CPU;
    double iStart, iElaps;
    iStart = timeCount();
    Sum_vec_on_CPU(h_a, h_b, CPU_res, DSIZE);
    iElaps = timeCount() - iStart;
    printf("Time taken on CPU : %f seconds.\n", iElaps);

    // allocate memory on device
    hipMalloc((void **)&d_a, nBytes);
    hipMalloc((void **)&d_b, nBytes);
    hipMalloc((void **)&d_c, nBytes);
    cudaCheckErrors("hipMalloc failure");

    // Copy memory from host to device
    hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, GPU_res, nBytes, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");
    // launch kernel
    // warming up
    dim3 block(512);
    dim3 grid((DSIZE + block.x -1)/ block.x); 
    iStart = timeCount();
    Sum_vec_on_GPU<<<grid, block>>>(d_a, d_b, d_c, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel during warming up : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch warming up failure");
    // Count time;
    // cudaEvent_t start, stop;
    // cudaEventCreate(&start);
    // cudaEventCreate(&stop);

    // Record the start event
    // cudaEventRecord(start, 0);  
    iStart = timeCount();
    Sum_vec_on_GPU<<<grid, block>>>(d_a, d_b, d_c, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel : %f seconds.\n", iElaps);
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    //  // Calculate the elapsed time in milliseconds
    // float milliseconds = 0;
    // cudaEventElapsedTime(&milliseconds, start, stop);
    // // Clean up
    // cudaEventDestroy(start);
    // cudaEventDestroy(stop);
    // // Print the time taken by the kernel execution
    // printf("Time taken by kernel: %f ms\n", milliseconds);
    cudaCheckErrors("kernel launch failure");
    // copy result from device to host
    hipMemcpy(GPU_res, d_c, nBytes, hipMemcpyDeviceToHost);
    cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
    // check result
    CheckResult(CPU_res, GPU_res, DSIZE);
    printf("Success!\n"); 
    free(h_a);
    free(h_b);
    free(CPU_res);
    free(GPU_res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

