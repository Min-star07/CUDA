#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char ** argv){
    //define total data element
    int nElem = 1024;
    // define grid and block structure
    dim3 block(1024);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("Grid.x = %d, block.x = %d\n", grid.x, block.x);
    // reset block
    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("Grid.x = %d, block.x = %d\n", grid.x, block.x);
    // reset block
    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("Grid.x = %d, block.x = %d\n", grid.x, block.x);
    // reset block
    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("Grid.x = %d, block.x = %d\n", grid.x, block.x);
    // reset device before you leave
    hipDeviceReset();  // Make sure no CUDA-related memory leaks occur.
    return 0;

}