#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
double timeCount(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec *1.e-6);
}
// Initilization data on host
void InitiData(float *vec, int size){
    for(int i =0; i < size; i++){
        vec[i] = rand()/(float)RAND_MAX;
        // printf("%d, %.1f\n", i, vec[i]);
    }
}

// CUDA ERROR CHECK
#define cudaCheckErrors(msg){\
    hipError_t __err = hipGetLastError();\
    if(__err != hipSuccess){\
    fprintf(stderr, "Fatal error: %s ===> %s ===>  %s : %d\n", msg, hipGetErrorString(__err), __FILE__, __LINE__);\
    fprintf(stderr, "FURTHER CHECK *****  ABORTION\n");\
    exit(1);\
};\
}
// sum on host
void Sum_vec_on_CPU(float * vec1, float sum, int size){
    sum = 0.0;
    for(int i = 0; i < size; i ++){
        sum += vec1[i];
    }
}
// kernal
__global__ void Sum_vec_on_GPU_without_shared(float *vec1, float* vec2, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
   

    int tid = threadIdx.x;
    float *idata = vec1 + blockDim.x * blockIdx.x;

    if(idx > size) return;
    
    for(int stride = 1; stride < blockDim.x; stride*=2){
        if(tid%(2*stride)==0){
            idata[tid] = idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) vec2[blockIdx.x] = idata[0]; 
}

// kernal
extern __shared__ float s_array[];                    // Shared memory array
__global__ void Sum_vec_on_GPU_with_shared(float *vec1, float* vec2, int size){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;  // Global index
    int stride = blockDim.x * gridDim.x;              // Stride for loop to cover all elements
    int tid = threadIdx.x;                            // Thread index within the block
    
    // Load elements into shared memory, with stride to handle larger arrays
    while(idx < size){
            s_array[tid] = vec1[idx];
            idx += stride;
            __syncthreads();
    }
    if(idx > size) return;
    
    float *idata = vec1 + blockDim.x * blockIdx.x;

    for(int stride = 1; stride < blockDim.x; stride*=2){
        if(tid%(2*stride)==0){
            idata[tid] = idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) vec2[blockIdx.x] = idata[0]; 
    
}

void setGPU(){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount  < 1){
        printf("No CUDA device found, exiting...\n");
        exit(1);
    }
    else{
        for(int i = 0; i < deviceCount; i++ ){
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
             printf("Device %d: %s has compute capability : Major: %d Minor: %d \n", i, deviceProp.name, deviceProp.major, deviceProp.minor);
            // set GPU
            hipSetDevice(i);
            printf("Set GPU %d.\n", i);
        }
    }
}

// compare the difference between CPU and GPU
void CheckResult(float CPU_res, float GPU_res, int size){
    float error_bar = 1e-6;
  
    if((CPU_res - GPU_res ) > error_bar){
        printf("ThE result is diffeence between GPU (%.1f) and CPU (%.1f)\n",  CPU_res,  GPU_res);
        exit(1);
    }
   
}

void sum_from_GPU(float *sum_part, float sum_all, int size){
    sum_all = 0;
    for(int i = 0; i < size; i++){
        sum_all += sum_part[i];
    }
}

int main(int argc, char** argv){
    // set GPU
    setGPU();
    cudaCheckErrors("set GPU failed");
    // define varilable on host and device
    float *A, CPU_res, *GPU_res_with_shared_memory, *GPU_res_without_shared_memory;
    const int DSIZE = 1<<24;
    size_t nBytes = DSIZE * sizeof(float);

    // warming up
    dim3 block(512);
    dim3 grid((DSIZE + block.x -1)/ block.x); 
    hipMallocManaged((void **)&A, nBytes);
    // cudaMallocManaged((void **)&CPU_res, sizeof(float));
    hipMallocManaged((void **)&GPU_res_with_shared_memory, grid.x* sizeof(float));
    hipMallocManaged((void **)&GPU_res_without_shared_memory, grid.x* sizeof(float));

    // initi data on host
    InitiData(A, DSIZE);

    // Count time;
    // Get the result oon the CPU;
    CPU_res = 0.0;
    double iStart, iElaps;
    iStart = timeCount();
    Sum_vec_on_CPU(A, CPU_res, DSIZE);
    iElaps = timeCount() - iStart;
    printf("Time taken on CPU : %f seconds.\n", iElaps);

    
    iStart = timeCount();
    Sum_vec_on_GPU_without_shared<<<grid, block>>>(A , GPU_res_with_shared_memory, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel during warming up : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch warming up failure");

    // launch kernel
    iStart = timeCount();
    Sum_vec_on_GPU_without_shared<<<grid, block>>>(A , GPU_res_without_shared_memory, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel without shared memory : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch failure");
    float GPU_res_without_shared_memory_sum =0;
    sum_from_GPU(GPU_res_without_shared_memory, GPU_res_without_shared_memory_sum, grid.x);
     // launch kernel
    iStart = timeCount();
    Sum_vec_on_GPU_with_shared<<<grid, block, block.x *sizeof(float)>>>(A , GPU_res_with_shared_memory, DSIZE);
    hipDeviceSynchronize();
    iElaps = timeCount() - iStart;
    printf("Time took on kernel with shared memory : %f seconds.\n", iElaps);
    cudaCheckErrors("kernel launch failure");
    float GPU_res_with_shared_memory_sum =0;
    sum_from_GPU(GPU_res_with_shared_memory, GPU_res_with_shared_memory_sum,grid.x);

    // check result
    CheckResult(CPU_res, GPU_res_without_shared_memory_sum, DSIZE);
    CheckResult(CPU_res, GPU_res_with_shared_memory_sum, DSIZE);
    printf("Success!\n"); 
    hipHostFree(A);
    // cudaFreeHost(CPU_res);
    hipHostFree(GPU_res_without_shared_memory);
    hipHostFree(GPU_res_with_shared_memory);

    return 0;
}


