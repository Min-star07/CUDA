#include <cstdio>
#include <hip/hip_runtime.h>

// CUDA error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            exit(1); \
        } \
    } while (0)

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 100;
    size_t size = N * sizeof(int);

    // Pinned host memory allocation
    int *h_a, *h_b, *h_c;
    hipHostMalloc(&h_a, size, hipHostMallocDefault);  // Pinned memory on the host
    hipHostMalloc(&h_b, size, hipHostMallocDefault);
    hipHostMalloc(&h_c, size, hipHostMallocDefault);

    // Initialize host arrays with values
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);  // Allocate memory on the GPU
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy to device failed");

    // Launch kernel for vector addition
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);
    cudaCheckErrors("Kernel launch failed");

    // Wait for kernel to complete before accessing results on the host
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel execution failed");

    // Copy result back from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy to host failed");

    // Print results
    for (int i = 0; i < N; ++i) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    cudaCheckErrors("hipFree failed");

    // Free pinned host memory
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}
